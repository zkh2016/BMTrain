
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <mutex>
#include <unordered_map>
#include <hipblaslt.h>

class CublasLtHandle {
public:
    static CublasLtHandle& getInstance() {
        static CublasLtHandle instance; // 唯一实例
        return instance;
    }

    hipblasLtHandle_t getHandle() {
        return handle;
    }

private:
    CublasLtHandle() {
        hipblasLtCreate(&handle);
    }

    ~CublasLtHandle() {
        hipblasLtDestroy(handle);
    }

    hipblasLtHandle_t handle;
};

class GemmEpilogueAlgoCache {
	public:
		size_t workspace_size = static_cast<size_t>(4) * 1024 * 1024;
		void* workspace;

		static GemmEpilogueAlgoCache& Instance() {
			static GemmEpilogueAlgoCache instance(30);
			return instance;
		}

		GemmEpilogueAlgoCache(GemmEpilogueAlgoCache const&) = delete;
		void operator=(GemmEpilogueAlgoCache const&) = delete;

		hipblasLtMatmulAlgo_t* GetGemmAlgo(hipblasLtHandle_t lt_handle,
				hipblasLtMatmulDesc_t op_desc,
				hipblasLtMatrixLayout_t a_desc,
				hipblasLtMatrixLayout_t b_desc,
				hipblasLtMatrixLayout_t c_desc,
				const void* alpha,
				const void* beta,
				const void* a,
				const void* b,
				void* c,
				hipStream_t stream){
				//void* workspace,
				//const size_t workspace_size) {
			if (search_times_ <= 0) return nullptr;

			int64_t seed = 0;
			std::hash<int64_t> hash_fn;

			HashMatmulDesc_(op_desc, &seed, hash_fn);
			HashMatrixLayoutDesc_(a_desc, &seed, hash_fn);
			HashMatrixLayoutDesc_(b_desc, &seed, hash_fn);
			HashMatrixLayoutDesc_(c_desc, &seed, hash_fn);

			hipblasLtMatmulAlgo_t ret;
			{
				std::lock_guard<std::mutex> lock(cache_mutex_);
				auto it = map_.find(seed);
				if (it != map_.end()) {
					return &(it->second);
				}
			}

			hipblasLtMatmulPreference_t preference;
			(
					hipblasLtMatmulPreferenceCreate(&preference));
			(
					hipblasLtMatmulPreferenceSetAttribute(
						preference,
						HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
						&workspace_size,
						sizeof(workspace_size)));

			int returned_results = 0;
			std::vector<hipblasLtMatmulHeuristicResult_t> heuristic_results(
					requested_algo_count_);
            auto heur_status = hipblasLtMatmulAlgoGetHeuristic(lt_handle,
                    op_desc,
                    a_desc,
                    b_desc,
                    c_desc,
                    c_desc,
                    preference,
                    requested_algo_count_,
                    heuristic_results.data(),
                    &returned_results);


			(
					hipblasLtMatmulPreferenceDestroy(preference));

			int best_algo_idx = -1;
			float best_algo_time = 0;

			// Run 100 times for warmup
			int warmup_algo_idx = 0;
			for (int t = 0; t < 100; t++) {
				hipblasStatus_t status =
					hipblasLtMatmul(lt_handle,
							op_desc,
							alpha,
							a,
							a_desc,
							b,
							b_desc,
							beta,
							c,
							c_desc,
							c,
							c_desc,
							&heuristic_results[warmup_algo_idx].algo,
							workspace,
							workspace_size,
							stream);
				if (status != HIPBLAS_STATUS_SUCCESS) {
					t = -1;
					warmup_algo_idx += 1;
					if (warmup_algo_idx == requested_algo_count_) {
						std::cout << "No GEMM epilogue algorithm support!" << status << std::endl;
					}
				}
			}

			hipEvent_t start_event, stop_event;
			(hipEventCreate(&start_event));
			(hipEventCreate(&stop_event));

			for (int algo_idx = 0; algo_idx < returned_results; ++algo_idx) {
				float curr_time = 0;
				for (int check_idx = 0; check_idx < search_times_; check_idx++) {
					float time = 0;
					(hipEventRecord(start_event, stream));

					hipblasStatus_t status =
						hipblasLtMatmul(lt_handle,
								op_desc,
								alpha,
								a,
								a_desc,
								b,
								b_desc,
								beta,
								c,
								c_desc,
								c,
								c_desc,
								&heuristic_results[algo_idx].algo,
								workspace,
								workspace_size,
								stream);

					(hipEventRecord(stop_event, stream));
					(hipEventSynchronize(stop_event));
					(
							hipEventElapsedTime(&time, start_event, stop_event));
					curr_time += time;
					if (status != HIPBLAS_STATUS_SUCCESS) {
						curr_time = 3.40282e+038;  // Max Value of float
						break;
					}
				}

				curr_time = curr_time / search_times_;
				if (curr_time < best_algo_time || algo_idx == 0) {
					best_algo_idx = algo_idx;
					best_algo_time = curr_time;
				}
			}

			(hipEventDestroy(start_event));
			(hipEventDestroy(stop_event));

			if (best_algo_idx == -1) {
				std::cout << "No GEMM epilogue algorithm support!\n";
			}

			ret = heuristic_results[best_algo_idx].algo;

			std::lock_guard<std::mutex> lock(cache_mutex_);
			auto& algo_in_map = map_[seed];
			algo_in_map = ret;
			return &algo_in_map;
		}

	private:
		explicit GemmEpilogueAlgoCache(int search_times)
			: search_times_(search_times) {
				map_.clear();
				hipError_t status = hipMalloc(&workspace, workspace_size);
				std::cout << "create GemmEpilogueAlgoCache " << status << std::endl;
		}
		std::unordered_map<int64_t, hipblasLtMatmulAlgo_t> map_;
		int search_times_;
		const int requested_algo_count_ = 10;
		std::mutex cache_mutex_;

		void HashMatmulDesc_(hipblasLtMatmulDesc_t desc,
				int64_t* seed,
				const std::hash<int64_t>& hash_fn) {
			size_t size_to_write;
			int trans_a, trans_b;
			uint32_t epilogue;

			(hipblasLtMatmulDescGetAttribute(
						desc,
						HIPBLASLT_MATMUL_DESC_TRANSA,
						&trans_a,
						sizeof(trans_a),
						&size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(trans_a));

			(hipblasLtMatmulDescGetAttribute(
						desc,
						HIPBLASLT_MATMUL_DESC_TRANSB,
						&trans_b,
						sizeof(trans_b),
						&size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(trans_b));

			(hipblasLtMatmulDescGetAttribute(
						desc,
						HIPBLASLT_MATMUL_DESC_EPILOGUE,
						&epilogue,
						sizeof(epilogue),
						&size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(epilogue));
		}

		void HashMatrixLayoutDesc_(hipblasLtMatrixLayout_t desc,
				int64_t* seed,
				const std::hash<int64_t>& hash_fn) {
			size_t size_to_write;
			uint32_t dtype;
			int32_t batch;
			uint64_t row, col;
			int64_t ld, batch_offset;

			(hipblasLtMatrixLayoutGetAttribute(
						desc,
						HIPBLASLT_MATRIX_LAYOUT_TYPE,
						&dtype,
						sizeof(dtype),
						&size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(dtype));

			(hipblasLtMatrixLayoutGetAttribute(
						desc,
						HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT,
						&batch,
						sizeof(batch),
						&size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(batch));

			(hipblasLtMatrixLayoutGetAttribute(
						desc, HIPBLASLT_MATRIX_LAYOUT_ROWS, &row, sizeof(row), &size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(row));

			(hipblasLtMatrixLayoutGetAttribute(
						desc, HIPBLASLT_MATRIX_LAYOUT_COLS, &col, sizeof(col), &size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(col));

			(hipblasLtMatrixLayoutGetAttribute(
						desc, HIPBLASLT_MATRIX_LAYOUT_LD, &ld, sizeof(ld), &size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(ld));

			(hipblasLtMatrixLayoutGetAttribute(
						desc,
						HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET,
						&batch_offset,
						sizeof(batch_offset),
						&size_to_write));
			HashValue_(seed, hash_fn, static_cast<int64_t>(batch_offset));
		}

		void HashValue_(int64_t* seed,
				const std::hash<int64_t>& hash_fn,
				int64_t value) {
			*seed ^= hash_fn(value) + 0x9e3779b9 + (*seed << 6) + (*seed >> 2);
		}
};

template<typename T, bool backward=false>
void gemm(const T* A, const T* B, const T* Bias, 
        const int M, const int K, const int N,
        const bool trans_a, const bool trans_b,
        T* C,
        hipStream_t stream){
    hipDataType dtype;
    hipblasLtMatmulDesc_t desc;
    if (std::is_same<T, float>::value){
        dtype = HIP_R_32F;
        hipblasLtMatmulDescCreate(&desc, HIPBLAS_COMPUTE_32F, dtype); 
    }else{
        dtype = HIP_R_16F;
        hipblasLtMatmulDescCreate(&desc, HIPBLAS_COMPUTE_16F, dtype); 
    }

    hipblasLtMatrixLayout_t a_layout, b_layout, c_layout;
    if (Bias != nullptr){
        if (backward){
            hipblasLtEpilogue_t fused_db = HIPBLASLT_EPILOGUE_BGRADB;
            hipblasLtMatmulDescSetAttribute(
                    desc,
                    HIPBLASLT_MATMUL_DESC_EPILOGUE,
                    &fused_db,
                    sizeof(fused_db));
        }else{
            hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
            hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
        }
        hipblasLtMatmulDescSetAttribute(
                desc,
                HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                &Bias,
                sizeof(Bias));
    }
    hipblasLtMatrixLayoutCreate(&a_layout, dtype, K, M, K);
    hipblasOperation_t transa = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transb = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(
                  desc, HIPBLASLT_MATMUL_DESC_TRANSB, &transa, sizeof(transa));
    hipblasLtMatmulDescSetAttribute(
                  desc, HIPBLASLT_MATMUL_DESC_TRANSA, &transb, sizeof(transb));
    if(trans_a){
        hipblasLtMatrixLayoutCreate(&a_layout, dtype, M, K, M);
    }else{
        hipblasLtMatrixLayoutCreate(&a_layout, dtype, K, M, K);
    }

    if(trans_b){
        hipblasLtMatrixLayoutCreate(&b_layout, dtype, K, N, K);
    }else{
        hipblasLtMatrixLayoutCreate(&b_layout, dtype, N, K, N);
    }
    hipblasLtMatrixLayoutCreate(&c_layout, dtype, N, M, N);

    CublasLtHandle& handleInstance = CublasLtHandle::getInstance();
    hipblasLtHandle_t cublas_handle = handleInstance.getHandle();
    if (dtype == HIP_R_32F){
        float alpha = 1, beta = 0;
        hipblasLtMatmul(
                cublas_handle,
                desc,
                &alpha,
                B, b_layout,
                A, a_layout,
                &beta,
                C, c_layout,
                C, c_layout, 
                0,
                NULL,
                0,
                stream);
    }else{
        half alpha = __float2half(1), beta = __float2half(0);
		auto algo = GemmEpilogueAlgoCache::Instance().GetGemmAlgo(cublas_handle,
				desc,
				b_layout,
				a_layout,
				c_layout,
				&alpha,
				&beta,
			 	B,	
				A,
				C,
				stream);

        hipblasLtMatmul(
                cublas_handle,
                desc,
                &alpha,
                B, b_layout,
                A, a_layout,
                &beta,
                C, c_layout,
                C, c_layout, 
                NULL,
                NULL,
                0,
                stream);
    }
    hipblasLtMatrixLayoutDestroy(a_layout);
    hipblasLtMatrixLayoutDestroy(b_layout);
    hipblasLtMatrixLayoutDestroy(c_layout);
    hipblasLtMatmulDescDestroy(desc);
}

void linear_launcher(std::uintptr_t x, 
        std::uintptr_t weight,
        std::uintptr_t bias,
        std::uintptr_t out,
        const int batch,
        const int in_features,
        const int out_features,
        const bool trans_a,
        const bool trans_b,
        std::uintptr_t stream){
    auto* x_ptr = reinterpret_cast<half*>(x);
    auto* weight_ptr = reinterpret_cast<half*>(weight);
    auto* bias_ptr = reinterpret_cast<half*>(bias);
    auto* out_ptr = reinterpret_cast<half*>(out);
    auto curr_stream = reinterpret_cast<hipStream_t>(stream);
    gemm<half>(x_ptr, weight_ptr, bias_ptr, 
            batch, in_features, out_features, 
            trans_a, trans_b, 
            out_ptr, curr_stream);
}

void linear_backward_launcher(std::uintptr_t x, 
        std::uintptr_t weight,
        std::uintptr_t bias,
        std::uintptr_t out,
        std::uintptr_t dout,
        std::uintptr_t dx,
        std::uintptr_t dweight,
        std::uintptr_t dbias,
        const int batch,
        const int in_features,
        const int out_features,
        const bool trans_a,
        const bool trans_b,
        std::uintptr_t stream){
    auto* x_ptr = reinterpret_cast<half*>(x);
    auto* weight_ptr = reinterpret_cast<half*>(weight);
    auto* bias_ptr = reinterpret_cast<half*>(bias);
    auto* out_ptr = reinterpret_cast<half*>(out);
    auto* dout_ptr = reinterpret_cast<half*>(dout);
    auto* dx_ptr = reinterpret_cast<half*>(dx);
    auto* dweight_ptr = reinterpret_cast<half*>(dweight);
    auto* dbias_ptr = reinterpret_cast<half*>(dbias);
    const int m = batch;
    const int k = in_features;
    const int n = out_features;
    auto curr_stream = reinterpret_cast<hipStream_t>(stream);
    //weight(n, k) in forward
    //dx(m, k) = dout(m, n) * weight(n, k)
    gemm<half, true>(dout_ptr, weight_ptr, nullptr, 
            m, n, k, 
            false, false, 
            dx_ptr, curr_stream);
    //dweight(n, k) = Trans(dout(m, n)) * x(m,k) 
    //fused dbias
    gemm<half, true>(dout_ptr, x_ptr, dbias_ptr, 
            n, m, k, 
            true, false, 
            dweight_ptr, curr_stream);
}
