
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <hipblaslt.h>

#include <stdio.h>
#define BM_CUBLAS_ASSERT(status) do{hipblasStatus_t v = (status);if (v != HIPBLAS_STATUS_SUCCESS) std::cout << "CUBLAS Error: " #status, __FILE__, __LINE__, __PRETTY_FUNCTION__, cublasGetErrorString(v);}while(0)


class CublasLtHandle {
public:
    static CublasLtHandle& getInstance() {
        static CublasLtHandle instance; // 唯一实例
        return instance;
    }

    hipblasLtHandle_t getHandle() {
        return handle;
    }

private:
    CublasLtHandle() {
        hipblasLtCreate(&handle);
    }

    ~CublasLtHandle() {
        hipblasLtDestroy(handle);
    }

    hipblasLtHandle_t handle;
};

template<typename T>
class Linear{
    uint32_t _in_features, _out_features;
    const T* _weight;
    const T* _bias;
    hipDataType _dtype;

    hipblasLtMatmulDesc_t _desc;
    hipblasLtMatrixLayout_t _weight_layout;
    
public:
    Linear(const uint32_t in_features, const uint32_t out_features, const T* weight, const T*bias, const int dtype)
        : _in_features(in_features),
        _out_features(out_features),
        _weight(weight),
        _bias(bias) {
        if(dtype == 0){ //fp16
            _dtype = HIP_R_16F;
            hipblasLtMatmulDescCreate(&_desc, HIPBLAS_COMPUTE_16F, HIP_R_16F); 
        }else if(dtype == 1) { //fp32
            _dtype = HIP_R_32F;
            hipblasLtMatmulDescCreate(&_desc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
        }
        hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
        hipblasLtMatmulDescSetAttribute(_desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
        hipblasLtMatrixLayoutCreate(&_weight_layout, _dtype, _out_features, _in_features, _out_features);
        auto status = hipblasLtMatmulDescSetAttribute(
                _desc,
                HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                &_bias,
                sizeof(_bias));
    }
    ~Linear(){
        hipblasLtMatmulDescDestroy(_desc);
        hipblasLtMatrixLayoutDestroy(_weight_layout);
    }
    void forward(
            const int batch, 
            const bool trans_a,
            const bool trans_b,
            const T* x, 
            T* out, 
            hipStream_t stream){
        hipblasLtMatrixLayout_t x_layout, out_layout;
        hipblasLtMatrixLayoutCreate(&x_layout, _dtype, _in_features, batch, _in_features);
        const int M = batch;
        const int K = _in_features;
        const int N = _out_features;
        hipblasOperation_t transa = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        hipblasOperation_t transb = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        hipblasLtMatmulDescSetAttribute(
                      _desc, HIPBLASLT_MATMUL_DESC_TRANSB, &transa, sizeof(transa));
        hipblasLtMatmulDescSetAttribute(
                      _desc, HIPBLASLT_MATMUL_DESC_TRANSA, &transb, sizeof(transb));
        if(trans_a){
            hipblasLtMatrixLayoutCreate(&x_layout, _dtype, M, K, M);
        }else{
            hipblasLtMatrixLayoutCreate(&x_layout, _dtype, K, M, K);
        }

        if(trans_b){
            hipblasLtMatrixLayoutCreate(&_weight_layout, _dtype, K, N, K);
        }else{
            hipblasLtMatrixLayoutCreate(&_weight_layout, _dtype, N, K, N);
        }
        hipblasLtMatrixLayoutCreate(&out_layout, _dtype, N, M, N);

        CublasLtHandle& handleInstance = CublasLtHandle::getInstance();
        hipblasLtHandle_t cublas_handle = handleInstance.getHandle();

        if (_dtype == HIP_R_32F){
            float alpha = 1, beta = 0;
            hipblasLtMatmul(
                    cublas_handle,
                    _desc,
                    &alpha,
                    _weight, _weight_layout,
                    x, x_layout,
                    &beta,
                    out, out_layout,
                    out, out_layout, 
                    0,
                    NULL,
                    0,
                    stream);
        }else{
            half alpha = __float2half(1), beta = __float2half(0);
            hipblasLtMatmul(
                    cublas_handle,
                    _desc,
                    &alpha,
                    _weight, _weight_layout,
                    x, x_layout,
                    &beta,
                    out, out_layout,
                    out, out_layout, 
                    NULL,
                    NULL,
                    0,
                    stream);
        }
        hipblasLtMatrixLayoutDestroy(x_layout);
        hipblasLtMatrixLayoutDestroy(out_layout);
    }
    void backward(const int batch, const T*x, const T* out, const T* out_grad, T*dx, T* dweight, T* dbias, hipStream_t stream){

    }
};

void linear_launcher(std::uintptr_t x, 
        std::uintptr_t weight,
        std::uintptr_t bias,
        std::uintptr_t out,
        const int batch,
        const int in_features,
        const int out_features,
        const bool trans_a,
        const bool trans_b,
        std::uintptr_t stream){
    auto* x_ptr = reinterpret_cast<half*>(x);
    auto* weight_ptr = reinterpret_cast<half*>(weight);
    auto* bias_ptr = reinterpret_cast<half*>(bias);
    auto* out_ptr = reinterpret_cast<half*>(out);
    Linear<half> linear(in_features, out_features, weight_ptr, bias_ptr, 0);
    auto curr_stream = reinterpret_cast<hipStream_t>(stream);
    linear.forward(batch, trans_a, trans_b, x_ptr, out_ptr, curr_stream);
}
